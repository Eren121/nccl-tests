#include <stdio.h>
#include <stdint.h>
#include <mpi.h>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define MPICHECK(cmd) do { \
    int e = cmd; \
    if (e != MPI_SUCCESS) { \
        fprintf(stderr, "MPI error %d at %s:%d\n", e, __FILE__, __LINE__); \
        MPI_Abort(MPI_COMM_WORLD, e); \
    } \
} while(0)

#define CUDACHECK(cmd) do { \
    hipError_t e = cmd; \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

int main(int argc, char *argv[]) {
    int size = 32 * 1024 * 1024;  // buffer size
    int myRank, nRanks, localRank = 0;

    // Initialize MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

    ncclUniqueId id;
    ncclComm_t comm;
    float *sendbuff, *recvbuff;
    hipStream_t s;

    // Rank 0 generates the NCCL unique ID
    if (myRank == 0) {
        ncclGetUniqueId(&id);
    }

    // Broadcast the NCCL unique ID to all ranks
    MPICHECK(MPI_Bcast(&id, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD));

    // Optional: print to confirm all ranks received it
    printf("Rank %d received NCCL unique ID\n", myRank);

    // Determine local GPU (here, simple mapping rank -> device)
    localRank = myRank; // for single node with 2 ranks
    CUDACHECK(hipSetDevice(0));

    // Allocate device buffers
    CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
    CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(&s));

    // Initialize NCCL communicator
    ncclCommInitRank(&comm, nRanks, id, myRank);

    // Now you can use NCCL to communicate between GPUs

    // Cleanup
    ncclCommDestroy(comm);
    hipFree(sendbuff);
    hipFree(recvbuff);
    hipStreamDestroy(s);
    MPICHECK(MPI_Finalize());

    return 0;
}